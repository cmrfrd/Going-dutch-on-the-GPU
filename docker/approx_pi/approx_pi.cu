#include <iostream>
#include <limits>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

using std::cout;
using std::endl;

typedef unsigned long long Count;
typedef std::numeric_limits<double> DblLim;

const Count WARP_SIZE = 32;
const Count NBLOCKS = 64;
const Count ITERATIONS = 10000000;
const Count REPETITIONS = 400;

__global__ void monte_carlo_pi(Count *totals) {

  // Create shared memory for block
	__shared__ Count counter[WARP_SIZE];
	counter[threadIdx.x] = 0;

	// Unique ID of the thread
  // use this id to seed the rng for each thread
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	hiprandState_t rng;
	hiprand_init(clock64(), tid, 0, &rng);

	// Run through iterations, sample two uniform points,
  // then calculate test if points fall within circle
	for (int i = 0; i < ITERATIONS; i++) {
		float x = hiprand_uniform(&rng);
		float y = hiprand_uniform(&rng);
		counter[threadIdx.x] += 1 - int(x * x + y * y);
	}

	// In every block use the first thread to aggregate the results
  // using the shared memory within the block. Shared memory is fast!
	if (threadIdx.x == 0) {
		totals[blockIdx.x] = 0;
		for (int i = 0; i < WARP_SIZE; i++) {
			totals[blockIdx.x] += counter[i];
		}
	}
}

int main(int argc, char **argv) {

  // Set precision of cout numbers
  cout.precision(DblLim::max_digits10);

  // Check if there is a cuda device available
	int numDev;
	hipGetDeviceCount(&numDev);
	if (numDev < 1) {
		cout << "CUDA device missing! Do you need to use optirun?\n";
		return 1;
	}

  // Log base params
	cout << "Starting monte carlo simulation with \n"
       << NBLOCKS << " blocks, \n"
       << WARP_SIZE << " threads, and \n"
       << ITERATIONS << " iterations, over \n"
       << REPETITIONS << " repetitions" << endl;

	// Allocate duplicate size host and device memory to store
  // the counts of each blocks monte carlo process
	Count *hostOutput, *deviceOutput;
	hostOutput = new Count[NBLOCKS]; // Host memory
	hipMalloc(&deviceOutput, sizeof(Count) * NBLOCKS); // Device memory

	Count total = 0;
  Count tests = NBLOCKS * ITERATIONS * WARP_SIZE;

  for (int repetition = 1; repetition <= REPETITIONS; repetition++) {

    // Launch kernel
    monte_carlo_pi<<<NBLOCKS, WARP_SIZE>>>(deviceOutput);

    // Copy back memory used on device and free
    hipMemcpy(hostOutput, deviceOutput, sizeof(Count) * NBLOCKS, hipMemcpyDeviceToHost);

    // Compute total hits
    for (int i = 0; i < NBLOCKS; i++) {
      total += hostOutput[i];
    }

    // Set maximum precision for decimal printing
    cout << "π ≅ " << 4.0 * (double)total/(double)(tests * repetition)
         << endl;
  }

  // Free device and host memory and exit process
	hipFree(deviceOutput);
  free(hostOutput);
	return 0;
}
